#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

// VectorAdd kernel
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < len; idx += blockDim.x * gridDim.x) {
        out[idx] = in1[idx] + in2[idx];
    }
}

int main(int argc, char *argv[]) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // print vector length and size
    int numElements = 10000;
    size_t vectorSize_inbytes = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // allocating host memory for input and output vectors on heap
    float *h_A = (float *) malloc(vectorSize_inbytes); // a
    float *h_B = (float *) malloc(vectorSize_inbytes); // b
    float *h_C = (float *) malloc(vectorSize_inbytes); // res
    // Verify CPU(host) allocation succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Create ptrs to GPU(device) memory
    float *d_A;
    float *d_B;
    float *d_C;
    // initialize input vectors a,b
    for (int i=0; i<numElements; ++i){
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate GPU memory using hipMalloc(addrOfthePtr, sizeOfallocatedMemory), with error checker
    // err = hipMalloc((void**) &d_A, vectorSize_inbytes);
    CHECK(hipMalloc((void**) &d_A, vectorSize_inbytes));
    // if (err != hipSuccess) // verify GPU allocation succeeded
    // {
    //     fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    err = hipMalloc((void**) &d_B, vectorSize_inbytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void**) &d_C, vectorSize_inbytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy host memory content to device memory using cudaMemCpy(p2dest, p2source, sizeofTransfer(in byte), direction)
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, vectorSize_inbytes, hipMemcpyHostToDevice); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    checkCudaErrors(hipMemcpy(d_B, h_B, vectorSize_inbytes, hipMemcpyHostToDevice)); 

    // Prep of launching CUDA kernel
    int blocksize = 256; // 256 threads per block by default
    if (argc > 1) blocksize = atoi(argv[1]); // When specified, blocksize=argv[1]
    int gridsize = 0; // ceil(nEle/blocksize)
    if (argc > 2) gridsize =  atoi(argv[2]);
    // printf("CUDA kernel launch with %d blocks of %d threads\n", gridsize, blocksize);
    // Initialize block and grid dim.
    dim3 DimGrid(gridsize, 1, 1);
    dim3 DimBlock(blocksize, 1, 1);
    
    // Launch kernel
    vecAdd<<<DimGrid, DimBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) // verify kernel excecution succeeded
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy res from device memory back to host memory using cudaMemCpy
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, vectorSize_inbytes, hipMemcpyDeviceToHost); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        // printf("expect: %f, ele: %f\n", h_A[i] + h_B[i], h_C[i]);
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stdout, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");


    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}